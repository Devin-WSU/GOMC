#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include<time.h>
#include<thrust/device_ptr.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/sort.h>
#include <ctime>
#include<hiprand/hiprand_kernel.h>
#include<hiprand.h>
#include<random>
using namespace std;
#define numOfArrays 10000
#define maxElements 1000
int main (){
    const int range_from = 0;
    const unsigned int range_to = 2147483647; //2^31 - 1
    random_device rand_dev;
    mt19937 generator(rand_dev());
    uniform_int_distribution<int> distr(range_from, range_to);
    thrust::host_vector<int> h_vec(numOfArrays*maxElements);
    thrust::device_vector<int> d_vec;
    thrust::host_vector<int> h_keys(numOfArrays*maxElements);
    thrust::device_vector<int> d_keys;
    srand(time(NULL));
    size_t f, t;
    hipSetDevice(0);
    hipMemGetInfo(&f, &t);
    //new data gens
    for(int i = 0; i < numOfArrays; i++){
        for(int j = 0; j < maxElements; j++){
            h_vec[i*maxElements+j] = distr(generator) ;
        }
    }
                
        //initializing the keys
    int timeKeys = clock();
    for(int i = 0; i < numOfArrays; i++){
    for(int j = 0; j < maxElements; j++){
        h_keys[i*maxElements+j] = i;
        }
    }
    timeKeys = clock()-timeKeys;
    //copying the data to device
    d_vec = h_vec;
    d_keys = h_keys;
    int start_s=clock();
    thrust::stable_sort(d_keys.begin(), d_keys.end());
    int stop_s=clock(); 
    //copying back
    h_vec = d_vec;
    cout  << ((stop_s-start_s)+timeKeys)/double(CLOCKS_PER_SEC)*1000 << endl;
    unsigned* my_device_pointer = thrust::raw_pointer_cast(&d_keys[0]);
    return 0;
}