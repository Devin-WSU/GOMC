
#include <hip/hip_runtime.h>
#ifdef GOMC_CUDA


#include "PrecisionChecker.cuh"

PrecisionChecker::PrecisionChecker(int i){}

void PrecisionChecker::sortCUDATuples(int * curr, int * neigh, double * val, int numberOfElements){

    thrust::device_vector< int > currVec(curr, curr+numberOfElements);
    thrust::device_vector< int > neighVec(neigh, neigh+numberOfElements);
    thrust::device_vector< double > valVec(val, val+numberOfElements);

    row_vec_dev_cuda_en = currVec;
    col_vec_dev_cuda_en = neighVec;
    val_vec_dev_cuda = valVec;

    A_first_cuda = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_cuda_en.begin(), col_vec_dev_cuda_en.begin(), val_vec_dev_cuda.begin()));
    A_last_cuda  = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_cuda_en.end(),   col_vec_dev_cuda_en.end(),   val_vec_dev_cuda.end()));

    thrust::sort(A_first_cuda, A_last_cuda, cmpEnergy());

    row_vec_cuda_en = row_vec_dev_cuda_en;
    col_vec_cuda_en = col_vec_dev_cuda_en;
    val_vec_cuda = val_vec_dev_cuda;
}

void PrecisionChecker::sortOMPTuples(int * curr, int * neigh, double * val, int numberOfElements){

    thrust::device_vector< int > currVec(curr, curr+numberOfElements);
    thrust::device_vector< int > neighVec(neigh, neigh+numberOfElements);
    thrust::device_vector< double > valVec(val, val+numberOfElements);

    row_vec_dev_omp_en = currVec;
    col_vec_dev_omp_en = neighVec;
    val_vec_dev_omp = valVec;

    A_first_omp = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_omp.begin(), col_vec_dev_omp.begin(), val_vec_dev_omp.begin()));
    A_last_omp  = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_omp.end(),   col_vec_dev_omp.end(),   val_vec_dev_omp.end()));

    thrust::sort(A_first_omp, A_last_omp, cmpEnergy());

    row_vec_omp_en = row_vec_dev_omp_en;
    col_vec_omp_en = col_vec_dev_omp_en;
    val_vec_omp = val_vec_dev_omp;
}
/*
void PrecisionChecker::sortCUDATuplesForce(int * curr, int * neigh, double * forceX, double * forceY, double * forceZ, int numberOfElements){

    thrust::device_vector< int > currVec(curr, curr+numberOfElements);
    thrust::device_vector< int > neighVec(neigh, neigh+numberOfElements);
    thrust::device_vector< double > valVecX(forceX, forceX+numberOfElements);
    thrust::device_vector< double > valVecY(forceY, forceY+numberOfElements);
    thrust::device_vector< double > valVecZ(forceZ, forceZ+numberOfElements);

    row_vec_dev_cuda = currVec;
    col_vec_dev_cuda = neighVec;
    valx_vec_dev_cuda = valVecX;
    valy_vec_dev_cuda = valVecY;
    valz_vec_dev_cuda = valVecZ;

    A_first_cuda_force = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_cuda.begin(), 
                                                                col_vec_dev_cuda.begin(), 
                                                                valx_vec_dev_cuda.begin(),
                                                                valy_vec_dev_cuda.begin(),
                                                                valz_vec_dev_cuda.begin()));

    A_last_cuda_force  = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_cuda.end(),   
                                                                col_vec_dev_cuda.end(),   
                                                                valx_vec_dev_cuda.end(),
                                                                valy_vec_dev_cuda.end(),
                                                                valz_vec_dev_cuda.end()));

    thrust::sort(A_first_cuda_force, A_last_cuda_force, cmpForce());

    row_vec_cuda = row_vec_dev_cuda;
    col_vec_cuda = col_vec_dev_cuda;
    valx_vec_cuda = valx_vec_dev_cuda;
    valy_vec_cuda = valy_vec_dev_cuda;
    valz_vec_cuda = valz_vec_dev_cuda;

}

void PrecisionChecker::sortOMPTuplesForce(int * curr, int * neigh, double * forceX, double * forceY, double * forceZ, int numberOfElements){

    thrust::device_vector< int > currVec(curr, curr+numberOfElements);
    thrust::device_vector< int > neighVec(neigh, neigh+numberOfElements);
    thrust::device_vector< double > valVecX(forceX, forceX+numberOfElements);
    thrust::device_vector< double > valVecY(forceY, forceY+numberOfElements);
    thrust::device_vector< double > valVecZ(forceZ, forceZ+numberOfElements);

    row_vec_dev_omp = currVec;
    col_vec_dev_omp = neighVec;
    valx_vec_dev_omp = valVecX;
    valy_vec_dev_omp = valVecY;
    valz_vec_dev_omp = valVecZ;

    A_first_omp_force = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_omp.begin(), 
                                                                col_vec_dev_omp.begin(), 
                                                                valx_vec_dev_omp.begin(),
                                                                valy_vec_dev_omp.begin(),
                                                                valz_vec_dev_omp.begin()));

    A_last_omp_force  = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_omp.end(),   
                                                                col_vec_dev_omp.end(),   
                                                                valx_vec_dev_omp.end(),
                                                                valy_vec_dev_omp.end(),
                                                                valz_vec_dev_omp.end()));

    thrust::sort(A_first_omp_force, A_last_omp_force, cmpForce());

    row_vec_omp = row_vec_dev_omp;
    col_vec_omp = col_vec_dev_omp;
    valx_vec_omp = valx_vec_dev_omp;
    valy_vec_omp = valy_vec_dev_omp;
    valz_vec_omp = valz_vec_dev_omp;
}
*/

bool PrecisionChecker::AlmostEqualUlps(float A, float B, int maxUlpsDiff)
{
    Float_t uA(A);
    Float_t uB(B);
    // Different signs means they do not match.
    if (uA.Negative() != uB.Negative())
    {
        // Check for equality to make sure +0==-0
        if (A == B)
            return true;
        return false;
    }
    // Find the difference in ULPs.
    int ulpsDiff = abs(uA.i - uB.i);
    if (ulpsDiff <= maxUlpsDiff)
        return true;
    return false;
}
bool PrecisionChecker::AlmostEqualUlps(double A, double B, int maxUlpsDiff)
{
    Double_t uA(A);
    Double_t uB(B);
    // Different signs means they do not match.
    if (uA.Negative() != uB.Negative())
    {
        // Check for equality to make sure +0==-0
        if (A == B)
            return true;
        return false;
    }
    // Find the difference in ULPs.
    int ulpsDiff = abs(uA.i - uB.i);
    if (ulpsDiff <= maxUlpsDiff)
        return true;
    return false;
}

#endif