#include "hip/hip_runtime.h"
#include "PrecisionChecker.cuh"

PrecisionChecker::PrecisionChecker(int i){}

void PrecisionChecker::sortCUDATuples(int * curr, int * neigh, double * val, int numberOfElements){

    thrust::device_vector< int > currVec(curr, curr+numberOfElements);
    thrust::device_vector< int > neighVec(neigh, neigh+numberOfElements);
    thrust::device_vector< double > valVec(val, val+numberOfElements);

    row_vec_dev_cuda = currVec;
    col_vec_dev_cuda = neighVec;
    val_vec_dev_cuda = valVec;

    A_first_cuda = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_cuda.begin(), col_vec_dev_cuda.begin(), val_vec_dev_cuda.begin()));
    A_last_cuda  = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_cuda.end(),   col_vec_dev_cuda.end(),   val_vec_dev_cuda.end()));

    thrust::sort(A_first_cuda, A_last_cuda, cmp());

    row_vec_cuda = row_vec_dev_cuda;
    col_vec_cuda = col_vec_dev_cuda;
    val_vec_cuda = val_vec_dev_cuda;
}

void PrecisionChecker::sortOMPTuples(int * curr, int * neigh, double * val, int numberOfElements){

    thrust::device_vector< int > currVec(curr, curr+numberOfElements);
    thrust::device_vector< int > neighVec(neigh, neigh+numberOfElements);
    thrust::device_vector< double > valVec(val, val+numberOfElements);

    row_vec_dev_omp = currVec;
    col_vec_dev_omp = neighVec;
    val_vec_dev_omp = valVec;

    A_first_omp = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_omp.begin(), col_vec_dev_omp.begin(), val_vec_dev_omp.begin()));
    A_last_omp  = thrust::make_zip_iterator(thrust::make_tuple(row_vec_dev_omp.end(),   col_vec_dev_omp.end(),   val_vec_dev_omp.end()));

    thrust::sort(A_first_omp, A_last_omp, cmp());

    row_vec_omp = row_vec_dev_omp;
    col_vec_omp = col_vec_dev_omp;
    val_vec_omp = val_vec_dev_omp;
}


bool PrecisionChecker::AlmostEqualUlps(float A, float B, int maxUlpsDiff)
{
    Float_t uA(A);
    Float_t uB(B);
    // Different signs means they do not match.
    if (uA.Negative() != uB.Negative())
    {
        // Check for equality to make sure +0==-0
        if (A == B)
            return true;
        return false;
    }
    // Find the difference in ULPs.
    int ulpsDiff = abs(uA.i - uB.i);
    if (ulpsDiff <= maxUlpsDiff)
        return true;
    return false;
}
bool PrecisionChecker::AlmostEqualUlps(double A, double B, int maxUlpsDiff)
{
    Double_t uA(A);
    Double_t uB(B);
    // Different signs means they do not match.
    if (uA.Negative() != uB.Negative())
    {
        // Check for equality to make sure +0==-0
        if (A == B)
            return true;
        return false;
    }
    // Find the difference in ULPs.
    int ulpsDiff = abs(uA.i - uB.i);
    if (ulpsDiff <= maxUlpsDiff)
        return true;
    return false;
}