
#include <hip/hip_runtime.h>
/*******************************************************************************
GPU OPTIMIZED MONTE CARLO (GOMC) 2.60
Copyright (C) 2018  GOMC Group
A copy of the GNU General Public License can be found in the COPYRIGHT.txt
along with this program, also can be found at <http://www.gnu.org/licenses/>.
********************************************************************************/
#ifdef GOMC_CUDA
#include <cuda.h>
#include "cub/cub.cuh"
#include <stdio.h>
#include "ConstantDefinitionsCUDAKernel.cuh"
#include "CalculateMinImageCUDAKernel.cuh"
#include "CalculateForceCUDAKernel.cuh"
#include "CalculateEnergyCUDAKernel.cuh"
#include "CUDAMemoryManager.cuh"
#include <vector>
#define NUMBER_OF_NEIGHBOR_CELL 27

using namespace cub;

struct Lock;

void CallBoxInterGPU(VariablesCUDA *vars,
                     std::vector<int> cellVector,
                     std::vector<int> cellStartIndex,
                     std::vector<std::vector<int> > neighborList,
                     XYZArray const &coords,
                     BoxDimensions const &boxAxes,
                     bool electrostatic,
                     std::vector<double> particleCharge,
                     std::vector<int> particleKind,
                     std::vector<int> particleMol,
                     double &REn,
                     double &LJEn,
                     bool sc_coul,
                     double sc_sigma_6,
                     double sc_alpha,
                     uint sc_power,
                     uint const box,
                     int * hostEnergyVectorLJKeys,
                     int * hostEnergyVectorREnKeys,
                     double * hostEnergyVectorLJValues,
                     double * hostEnergyVectorREnValues,
                     uint * numberOfInters
                    )
{
  int atomNumber = coords.Count();
  int cellVectorCount = cellVector.size();
  int neighborListCount = neighborList.size() * NUMBER_OF_NEIGHBOR_CELL;
  int numberOfCells = neighborList.size();
  int *gpu_particleKind, *gpu_particleMol;
  int *gpu_neighborList, *gpu_cellStartIndex;
  int blocksPerGrid, threadsPerBlock;
  int energyVectorLen = 0;
  double *gpu_particleCharge;
  double *gpu_REn, *gpu_LJEn;
  double *gpu_final_REn, *gpu_final_LJEn;

  int *gpu_REnForSortAndCPUReductionKeys, *gpu_LJEnForSortAndCPUReductionKeys;
  double *gpu_REnForSortAndCPUReductionValues, *gpu_LJEnForSortAndCPUReductionValues, *gpu_REnSortedValues, *gpu_LJEnSortedValues;



  // Run the kernel
  threadsPerBlock = 256;
  blocksPerGrid = (int)(numberOfCells * NUMBER_OF_NEIGHBOR_CELL);
  energyVectorLen = numberOfCells * NUMBER_OF_NEIGHBOR_CELL * threadsPerBlock;

  // Convert neighbor list to 1D array
  std::vector<int> neighborlist1D(neighborListCount);
  for(int i = 0; i < neighborList.size(); i++) {
    for(int j = 0; j < NUMBER_OF_NEIGHBOR_CELL; j++) {
      neighborlist1D[i * NUMBER_OF_NEIGHBOR_CELL + j] = neighborList[i][j];
    }
  }


  CUMALLOC((void**) &gpu_REnForSortAndCPUReductionKeys, energyVectorLen * sizeof(int));
  CUMALLOC((void**) &gpu_LJEnForSortAndCPUReductionKeys, energyVectorLen * sizeof(int));
  CUMALLOC((void**) &gpu_REnForSortAndCPUReductionValues, energyVectorLen * sizeof(double));
  CUMALLOC((void**) &gpu_LJEnForSortAndCPUReductionValues, energyVectorLen * sizeof(double));

  CUMALLOC((void**) &gpu_REnSortedValues, energyVectorLen * sizeof(double));
  CUMALLOC((void**) &gpu_LJEnSortedValues, energyVectorLen * sizeof(double));

  CUMALLOC((void**) &gpu_neighborList, neighborListCount * sizeof(int));
  CUMALLOC((void**) &gpu_cellStartIndex, cellStartIndex.size() * sizeof(int));
  CUMALLOC((void**) &gpu_particleCharge, particleCharge.size() * sizeof(double));
  CUMALLOC((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
  CUMALLOC((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
  CUMALLOC((void**) &gpu_REn, energyVectorLen * sizeof(double));
  CUMALLOC((void**) &gpu_LJEn, energyVectorLen * sizeof(double));
  CUMALLOC((void**) &gpu_final_REn, sizeof(double));
  CUMALLOC((void**) &gpu_final_LJEn, sizeof(double));

  // Copy necessary data to GPU
  cudaMemcpy(gpu_neighborList, &neighborlist1D[0], neighborListCount * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_cellStartIndex, &cellStartIndex[0], cellStartIndex.size() * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_cellVector, &cellVector[0], atomNumber * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleCharge, &particleCharge[0], particleCharge.size() * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleKind, &particleKind[0], particleKind.size() * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_particleMol, &particleMol[0], particleMol.size() * sizeof(int), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double), cudaMemcpyHostToDevice);
  cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double), cudaMemcpyHostToDevice);

  uint * numIntersPerCell = new uint[blocksPerGrid];
  uint * gpu_numIntersPerCell;
  CUMALLOC((void**) &gpu_numIntersPerCell, blocksPerGrid * sizeof(uint));
  
  CalcNumberOfInteractions <<< blocksPerGrid, threadsPerBlock>>>(gpu_cellStartIndex,
    vars->gpu_cellVector,
    gpu_neighborList,
    numberOfCells,
    cellVectorCount,
    vars->gpu_x,
    vars->gpu_y,
    vars->gpu_z,
    boxAxes.GetAxis(box).x,
    boxAxes.GetAxis(box).y,
    boxAxes.GetAxis(box).z,
    electrostatic,
    gpu_particleCharge,
    gpu_particleKind,
    gpu_particleMol,
    gpu_REn,
    gpu_LJEn,
    vars->gpu_sigmaSq,
    vars->gpu_epsilon_Cn,
    vars->gpu_n,
    vars->gpu_VDW_Kind,
    vars->gpu_isMartini,
    vars->gpu_count,
    vars->gpu_rCut,
    vars->gpu_rCutCoulomb,
    vars->gpu_rCutLow,
    vars->gpu_rOn,
    vars->gpu_alpha,
    vars->gpu_ewald,
    vars->gpu_diElectric_1,
    vars->gpu_nonOrth,
    vars->gpu_cell_x[box],
    vars->gpu_cell_y[box],
    vars->gpu_cell_z[box],
    vars->gpu_Invcell_x[box],
    vars->gpu_Invcell_y[box],
    vars->gpu_Invcell_z[box],
    sc_coul,
    sc_sigma_6,
    sc_alpha,
    sc_power,
    vars->gpu_rMin,
    vars->gpu_rMaxSq,
    vars->gpu_expConst,
    vars->gpu_molIndex,
    vars->gpu_kindIndex,
    vars->gpu_lambdaVDW,
    vars->gpu_lambdaCoulomb,
    vars->gpu_isFraction,
    box,
    gpu_numIntersPerCell);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);

  CubDebugExit(cudaMemcpy(numIntersPerCell, gpu_numIntersPerCell, blocksPerGrid * sizeof(uint),
  cudaMemcpyDeviceToHost));


  uint totalNumInters = 0;
  for (int i = 0; i < blocksPerGrid; i++){
    totalNumInters+=numIntersPerCell[i];
  }  
  CUMALLOC((void**) &vars->numberOfInters, sizeof(uint));
  // Set this for the flattened force calc
  CubDebugExit(cudaMemcpy(vars->numberOfInters, &totalNumInters, sizeof(uint), cudaMemcpyHostToDevice));

/*
  for (int i = 0; i < blocksPerGrid; i++){
    std::cout << "numIntersPerCell[" << i << "] : " << numIntersPerCell[i] << std::endl;
  }
*/
  uint tmp = 0;
  for (int i = 1; i <= blocksPerGrid; i++){
    for (int j = i-1; j < i; j++){
      numIntersPerCell[j] = tmp;
      tmp += numIntersPerCell[i];
    }
  }
/*
  std::cout << "number of interactions : " << totalNumInters << std::endl;
  for (int i = 0; i < blocksPerGrid; i++){
    std::cout << "numIntersPerCell[" << i << "] : " << numIntersPerCell[i] << std::endl;
  }
*/
  double * flatgpu_REn;
  double * flatgpu_LJEn;
  CUMALLOC((void**) &flatgpu_REn, totalNumInters * sizeof(double));
  CUMALLOC((void**) &flatgpu_LJEn, totalNumInters * sizeof(double));

  hostEnergyVectorLJKeys = (int*) malloc( sizeof(int) * totalNumInters);  
  hostEnergyVectorREnKeys = (int*) malloc( sizeof(int) * totalNumInters);
  
  int flatIndexREn = -1;
  int flatIndexLJEn = -1;

  int * gpu_flatIndexREn;
  int * gpu_flatIndexLJEn;
  CUMALLOC((void**) &gpu_flatIndexREn, sizeof(int));
  CUMALLOC((void**) &gpu_flatIndexLJEn, sizeof(int));

  CubDebugExit(cudaMemcpy(gpu_flatIndexREn, &flatIndexREn, sizeof(int), cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_flatIndexLJEn, &flatIndexLJEn, sizeof(int), cudaMemcpyHostToDevice));

  BoxInterGPUFlattened <<< blocksPerGrid, threadsPerBlock>>>(gpu_cellStartIndex,
    vars->gpu_cellVector,
    gpu_neighborList,
    numberOfCells,
    cellVectorCount,
    vars->gpu_x,
    vars->gpu_y,
    vars->gpu_z,
    boxAxes.GetAxis(box).x,
    boxAxes.GetAxis(box).y,
    boxAxes.GetAxis(box).z,
    electrostatic,
    gpu_particleCharge,
    gpu_particleKind,
    gpu_particleMol,
    gpu_REn,
    gpu_LJEn,
    vars->gpu_sigmaSq,
    vars->gpu_epsilon_Cn,
    vars->gpu_n,
    vars->gpu_VDW_Kind,
    vars->gpu_isMartini,
    vars->gpu_count,
    vars->gpu_rCut,
    vars->gpu_rCutCoulomb,
    vars->gpu_rCutLow,
    vars->gpu_rOn,
    vars->gpu_alpha,
    vars->gpu_ewald,
    vars->gpu_diElectric_1,
    vars->gpu_nonOrth,
    vars->gpu_cell_x[box],
    vars->gpu_cell_y[box],
    vars->gpu_cell_z[box],
    vars->gpu_Invcell_x[box],
    vars->gpu_Invcell_y[box],
    vars->gpu_Invcell_z[box],
    sc_coul,
    sc_sigma_6,
    sc_alpha,
    sc_power,
    vars->gpu_rMin,
    vars->gpu_rMaxSq,
    vars->gpu_expConst,
    vars->gpu_molIndex,
    vars->gpu_kindIndex,
    vars->gpu_lambdaVDW,
    vars->gpu_lambdaCoulomb,
    vars->gpu_isFraction,
    box,
    gpu_numIntersPerCell,
    flatgpu_REn,
    flatgpu_LJEn,
    gpu_flatIndexREn,
    gpu_flatIndexLJEn);
cudaDeviceSynchronize();
checkLastErrorCUDA(__FILE__, __LINE__);



/*
  BoxInterGPU <<< blocksPerGrid, threadsPerBlock>>>(gpu_cellStartIndex,
      vars->gpu_cellVector,
      gpu_neighborList,
      numberOfCells,
      cellVectorCount,
      vars->gpu_x,
      vars->gpu_y,
      vars->gpu_z,
      boxAxes.GetAxis(box).x,
      boxAxes.GetAxis(box).y,
      boxAxes.GetAxis(box).z,
      electrostatic,
      gpu_particleCharge,
      gpu_particleKind,
      gpu_particleMol,
      gpu_REn,
      gpu_LJEn,
      vars->gpu_sigmaSq,
      vars->gpu_epsilon_Cn,
      vars->gpu_n,
      vars->gpu_VDW_Kind,
      vars->gpu_isMartini,
      vars->gpu_count,
      vars->gpu_rCut,
      vars->gpu_rCutCoulomb,
      vars->gpu_rCutLow,
      vars->gpu_rOn,
      vars->gpu_alpha,
      vars->gpu_ewald,
      vars->gpu_diElectric_1,
      vars->gpu_nonOrth,
      vars->gpu_cell_x[box],
      vars->gpu_cell_y[box],
      vars->gpu_cell_z[box],
      vars->gpu_Invcell_x[box],
      vars->gpu_Invcell_y[box],
      vars->gpu_Invcell_z[box],
      sc_coul,
      sc_sigma_6,
      sc_alpha,
      sc_power,
      vars->gpu_rMin,
      vars->gpu_rMaxSq,
      vars->gpu_expConst,
      vars->gpu_molIndex,
      vars->gpu_kindIndex,
      vars->gpu_lambdaVDW,
      vars->gpu_lambdaCoulomb,
      vars->gpu_isFraction,
      box);
  cudaDeviceSynchronize();
  checkLastErrorCUDA(__FILE__, __LINE__);
*/
  // Copy Keys to device
  /*
  CubDebugExit(cudaMemcpy(gpu_REnForSortAndCPUReductionKeys, hostEnergyVectorREnKeys, energyVectorLen * sizeof(int),
                          cudaMemcpyHostToDevice));
  CubDebugExit(cudaMemcpy(gpu_LJEnForSortAndCPUReductionKeys, hostEnergyVectorLJKeys, energyVectorLen * sizeof(int),
                          cudaMemcpyHostToDevice));
*/
/*
  // Clone energy vector for sorting
  CubDebugExit(cudaMemcpy(gpu_REnForSortAndCPUReductionValues, flatgpu_REn, totalNumInters * sizeof(double),
                          cudaMemcpyDeviceToDevice));
  CubDebugExit(cudaMemcpy(gpu_LJEnForSortAndCPUReductionValues, flatgpu_LJEn, totalNumInters * sizeof(double),
                          cudaMemcpyDeviceToDevice));

  // Determine temporary device storage requirements
void     *d_temp_storage1 = NULL;
size_t   temp_storage_bytes1 = 0;
cub::DeviceRadixSort::SortPairs(d_temp_storage1, temp_storage_bytes1,
  gpu_REnForSortAndCPUReductionKeys, gpu_REnForSortAndCPUReductionKeys, gpu_REnForSortAndCPUReductionValues, gpu_REnSortedValues, energyVectorLen);
// Allocate temporary storage
cudaMalloc(&d_temp_storage1, temp_storage_bytes1);
// Run sorting operation
cub::DeviceRadixSort::SortPairs(d_temp_storage1, temp_storage_bytes1,
  gpu_REnForSortAndCPUReductionKeys, gpu_REnForSortAndCPUReductionKeys, gpu_REnForSortAndCPUReductionValues, gpu_REnSortedValues, energyVectorLen);

// Copy sorted Keys to back to host
CubDebugExit(cudaMemcpy(hostEnergyVectorREnKeys, gpu_REnForSortAndCPUReductionKeys, totalNumInters * sizeof(int),
    cudaMemcpyDeviceToHost));
CubDebugExit(cudaMemcpy(hostEnergyVectorLJKeys, gpu_LJEnForSortAndCPUReductionKeys, totalNumInters * sizeof(int),
    cudaMemcpyDeviceToHost));

*/
// Copy sorted Values back to host
CubDebugExit(cudaMemcpy(hostEnergyVectorREnValues, flatgpu_REn, totalNumInters * sizeof(double),
    cudaMemcpyDeviceToHost));
CubDebugExit(cudaMemcpy(hostEnergyVectorLJValues, flatgpu_LJEn, totalNumInters * sizeof(double),
    cudaMemcpyDeviceToHost));

  *numberOfInters = totalNumInters;

  // ReduceSum
  void * d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_REn,
                    gpu_final_REn, energyVectorLen);
  CubDebugExit(CUMALLOC(&d_temp_storage, temp_storage_bytes));
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_REn,
                    gpu_final_REn, energyVectorLen);
  CUFREE(d_temp_storage);

  // LJ ReduceSum
  d_temp_storage = NULL;
  temp_storage_bytes = 0;
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_LJEn,
                    gpu_final_LJEn, energyVectorLen);
  CubDebugExit(CUMALLOC(&d_temp_storage, temp_storage_bytes));
  DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, gpu_LJEn,
                    gpu_final_LJEn, energyVectorLen);
  CUFREE(d_temp_storage);
  // Copy back the result to CPU ! :)
  CubDebugExit(cudaMemcpy(&REn, gpu_final_REn, sizeof(double),
                          cudaMemcpyDeviceToHost));
  CubDebugExit(cudaMemcpy(&LJEn, gpu_final_LJEn, sizeof(double),
                          cudaMemcpyDeviceToHost));                        
/*
  std::cout <<   "gpu's box : " << box << std::endl;
       
  double cutoff = fmax(vars->gpu_rCut[0], vars->gpu_rCutCoulomb[0]);
  std::cout <<   "gpu's cutoff : " << cutoff << std::endl;
  std::cout <<   "gpu's gpu_rCut[0] : " << vars->gpu_rCut << std::endl;
  std::cout <<   "gpu's gpu_rCutCoulomb[box] : " << vars->gpu_rCutCoulomb << std::endl;*/

  CUFREE(gpu_particleCharge);
  CUFREE(gpu_particleKind);
  CUFREE(gpu_particleMol);
  CUFREE(gpu_REn);
  CUFREE(gpu_LJEn);
  CUFREE(gpu_final_REn);
  CUFREE(gpu_final_LJEn);
  CUFREE(gpu_neighborList);
  CUFREE(gpu_cellStartIndex);


  CUFREE(gpu_REnForSortAndCPUReductionKeys);
  CUFREE(gpu_LJEnForSortAndCPUReductionKeys);
  CUFREE(gpu_REnForSortAndCPUReductionValues);
  CUFREE(gpu_LJEnForSortAndCPUReductionValues);
  CUFREE(gpu_REnSortedValues);
  CUFREE(gpu_LJEnSortedValues);
}

void GetNumberOfInters(VariablesCUDA *vars,
  std::vector<int> cellVector,
  std::vector<int> cellStartIndex,
  std::vector<std::vector<int> > neighborList,
  XYZArray const &coords,
  BoxDimensions const &boxAxes,
  bool electrostatic,
  std::vector<double> particleCharge,
  std::vector<int> particleKind,
  std::vector<int> particleMol,
  double &REn,
  double &LJEn,
  bool sc_coul,
  double sc_sigma_6,
  double sc_alpha,
  uint sc_power,
  uint const box,
  uint * numberOfInters
 )
{
int atomNumber = coords.Count();
int cellVectorCount = cellVector.size();
int neighborListCount = neighborList.size() * NUMBER_OF_NEIGHBOR_CELL;
int numberOfCells = neighborList.size();
int *gpu_particleKind, *gpu_particleMol;
int *gpu_neighborList, *gpu_cellStartIndex;
int blocksPerGrid, threadsPerBlock;
int energyVectorLen = 0;
double *gpu_particleCharge;
double *gpu_REn, *gpu_LJEn;
double *gpu_final_REn, *gpu_final_LJEn;

int *gpu_REnForSortAndCPUReductionKeys, *gpu_LJEnForSortAndCPUReductionKeys;
double *gpu_REnForSortAndCPUReductionValues, *gpu_LJEnForSortAndCPUReductionValues, *gpu_REnSortedValues, *gpu_LJEnSortedValues;



// Run the kernel
threadsPerBlock = 256;
blocksPerGrid = (int)(numberOfCells * NUMBER_OF_NEIGHBOR_CELL);
energyVectorLen = numberOfCells * NUMBER_OF_NEIGHBOR_CELL * threadsPerBlock;

// Convert neighbor list to 1D array
std::vector<int> neighborlist1D(neighborListCount);
for(int i = 0; i < neighborList.size(); i++) {
for(int j = 0; j < NUMBER_OF_NEIGHBOR_CELL; j++) {
neighborlist1D[i * NUMBER_OF_NEIGHBOR_CELL + j] = neighborList[i][j];
}
}


CUMALLOC((void**) &gpu_REnForSortAndCPUReductionKeys, energyVectorLen * sizeof(int));
CUMALLOC((void**) &gpu_LJEnForSortAndCPUReductionKeys, energyVectorLen * sizeof(int));
CUMALLOC((void**) &gpu_REnForSortAndCPUReductionValues, energyVectorLen * sizeof(double));
CUMALLOC((void**) &gpu_LJEnForSortAndCPUReductionValues, energyVectorLen * sizeof(double));

CUMALLOC((void**) &gpu_REnSortedValues, energyVectorLen * sizeof(double));
CUMALLOC((void**) &gpu_LJEnSortedValues, energyVectorLen * sizeof(double));

CUMALLOC((void**) &gpu_neighborList, neighborListCount * sizeof(int));
CUMALLOC((void**) &gpu_cellStartIndex, cellStartIndex.size() * sizeof(int));
CUMALLOC((void**) &gpu_particleCharge, particleCharge.size() * sizeof(double));
CUMALLOC((void**) &gpu_particleKind, particleKind.size() * sizeof(int));
CUMALLOC((void**) &gpu_particleMol, particleMol.size() * sizeof(int));
CUMALLOC((void**) &gpu_REn, energyVectorLen * sizeof(double));
CUMALLOC((void**) &gpu_LJEn, energyVectorLen * sizeof(double));
CUMALLOC((void**) &gpu_final_REn, sizeof(double));
CUMALLOC((void**) &gpu_final_LJEn, sizeof(double));

// Copy necessary data to GPU
cudaMemcpy(gpu_neighborList, &neighborlist1D[0], neighborListCount * sizeof(int), cudaMemcpyHostToDevice);
cudaMemcpy(gpu_cellStartIndex, &cellStartIndex[0], cellStartIndex.size() * sizeof(int), cudaMemcpyHostToDevice);
cudaMemcpy(vars->gpu_cellVector, &cellVector[0], atomNumber * sizeof(int), cudaMemcpyHostToDevice);
cudaMemcpy(gpu_particleCharge, &particleCharge[0], particleCharge.size() * sizeof(double), cudaMemcpyHostToDevice);
cudaMemcpy(gpu_particleKind, &particleKind[0], particleKind.size() * sizeof(int), cudaMemcpyHostToDevice);
cudaMemcpy(gpu_particleMol, &particleMol[0], particleMol.size() * sizeof(int), cudaMemcpyHostToDevice);
cudaMemcpy(vars->gpu_x, coords.x, atomNumber * sizeof(double), cudaMemcpyHostToDevice);
cudaMemcpy(vars->gpu_y, coords.y, atomNumber * sizeof(double), cudaMemcpyHostToDevice);
cudaMemcpy(vars->gpu_z, coords.z, atomNumber * sizeof(double), cudaMemcpyHostToDevice);

uint * numIntersPerCell = new uint[blocksPerGrid];
uint * gpu_numIntersPerCell;
CUMALLOC((void**) &gpu_numIntersPerCell, blocksPerGrid * sizeof(uint));

CalcNumberOfInteractions <<< blocksPerGrid, threadsPerBlock>>>(gpu_cellStartIndex,
vars->gpu_cellVector,
gpu_neighborList,
numberOfCells,
cellVectorCount,
vars->gpu_x,
vars->gpu_y,
vars->gpu_z,
boxAxes.GetAxis(box).x,
boxAxes.GetAxis(box).y,
boxAxes.GetAxis(box).z,
electrostatic,
gpu_particleCharge,
gpu_particleKind,
gpu_particleMol,
gpu_REn,
gpu_LJEn,
vars->gpu_sigmaSq,
vars->gpu_epsilon_Cn,
vars->gpu_n,
vars->gpu_VDW_Kind,
vars->gpu_isMartini,
vars->gpu_count,
vars->gpu_rCut,
vars->gpu_rCutCoulomb,
vars->gpu_rCutLow,
vars->gpu_rOn,
vars->gpu_alpha,
vars->gpu_ewald,
vars->gpu_diElectric_1,
vars->gpu_nonOrth,
vars->gpu_cell_x[box],
vars->gpu_cell_y[box],
vars->gpu_cell_z[box],
vars->gpu_Invcell_x[box],
vars->gpu_Invcell_y[box],
vars->gpu_Invcell_z[box],
sc_coul,
sc_sigma_6,
sc_alpha,
sc_power,
vars->gpu_rMin,
vars->gpu_rMaxSq,
vars->gpu_expConst,
vars->gpu_molIndex,
vars->gpu_kindIndex,
vars->gpu_lambdaVDW,
vars->gpu_lambdaCoulomb,
vars->gpu_isFraction,
box,
gpu_numIntersPerCell);
cudaDeviceSynchronize();
checkLastErrorCUDA(__FILE__, __LINE__);

CubDebugExit(cudaMemcpy(numIntersPerCell, gpu_numIntersPerCell, blocksPerGrid * sizeof(uint),
cudaMemcpyDeviceToHost));


uint totalNumInters = 0;
for (int i = 0; i < blocksPerGrid; i++){
totalNumInters+=numIntersPerCell[i];
}  

*numberOfInters = totalNumInters;

CUFREE(gpu_particleCharge);
CUFREE(gpu_particleKind);
CUFREE(gpu_particleMol);
CUFREE(gpu_REn);
CUFREE(gpu_LJEn);
CUFREE(gpu_final_REn);
CUFREE(gpu_final_LJEn);
CUFREE(gpu_neighborList);
CUFREE(gpu_cellStartIndex);


CUFREE(gpu_REnForSortAndCPUReductionKeys);
CUFREE(gpu_LJEnForSortAndCPUReductionKeys);
CUFREE(gpu_REnForSortAndCPUReductionValues);
CUFREE(gpu_LJEnForSortAndCPUReductionValues);
CUFREE(gpu_REnSortedValues);
CUFREE(gpu_LJEnSortedValues);
}


__global__ void CalcNumberOfInteractions( int *gpu_cellStartIndex,
                                          int *gpu_cellVector,
                                          int *gpu_neighborList,
                                          int numberOfCells,
                                          int cellVectorCount,
                                          double *gpu_x,
                                          double *gpu_y,
                                          double *gpu_z,
                                          double xAxes,
                                          double yAxes,
                                          double zAxes,
                                          bool electrostatic,
                                          double *gpu_particleCharge,
                                          int *gpu_particleKind,
                                          int *gpu_particleMol,
                                          double *gpu_REn,
                                          double *gpu_LJEn,
                                          double *gpu_sigmaSq,
                                          double *gpu_epsilon_Cn,
                                          double *gpu_n,
                                          int *gpu_VDW_Kind,
                                          int *gpu_isMartini,
                                          int *gpu_count,
                                          double *gpu_rCut,
                                          double *gpu_rCutCoulomb,
                                          double *gpu_rCutLow,
                                          double *gpu_rOn,
                                          double *gpu_alpha,
                                          int *gpu_ewald,
                                          double *gpu_diElectric_1,
                                          int *gpu_nonOrth,
                                          double *gpu_cell_x,
                                          double *gpu_cell_y,
                                          double *gpu_cell_z,
                                          double *gpu_Invcell_x,
                                          double *gpu_Invcell_y,
                                          double *gpu_Invcell_z,
                                          bool sc_coul,
                                          double sc_sigma_6,
                                          double sc_alpha,
                                          uint sc_power,
                                          double *gpu_rMin,
                                          double *gpu_rMaxSq,
                                          double *gpu_expConst,
                                          int *gpu_molIndex,
                                          int *gpu_kindIndex,
                                          double *gpu_lambdaVDW,
                                          double *gpu_lambdaCoulomb,
                                          bool *gpu_isFraction,
                                          int box,
                                          uint * numIntersPerCell)
{

  // GJS For flattening //
  const int threadsPerBlock = 256;
  __shared__ uint cache[threadsPerBlock];
  uint myCounter = 0;

  double distSq;
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  gpu_REn[threadID] = 0.0;
  gpu_LJEn[threadID] = 0.0;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);

  int currentCell = blockIdx.x / 27;
  int nCellIndex = blockIdx.x;
  int neighborCell = gpu_neighborList[nCellIndex];

  // calculate number of particles inside neighbor Cell
  int particlesInsideCurrentCell, particlesInsideNeighboringCells;
  int endIndex = neighborCell != numberOfCells - 1 ?
                 gpu_cellStartIndex[neighborCell + 1] : cellVectorCount;
  particlesInsideNeighboringCells = endIndex - gpu_cellStartIndex[neighborCell];

  // Calculate number of particles inside current Cell
  endIndex = currentCell != numberOfCells - 1 ?
             gpu_cellStartIndex[currentCell + 1] : cellVectorCount;
  particlesInsideCurrentCell = endIndex - gpu_cellStartIndex[currentCell];

  // total number of pairs
  int numberOfPairs = particlesInsideCurrentCell * particlesInsideNeighboringCells;

  for(int pairIndex = threadIdx.x; pairIndex < numberOfPairs; pairIndex += blockDim.x) {
    int neighborParticleIndex = pairIndex / particlesInsideCurrentCell;
    int currentParticleIndex = pairIndex % particlesInsideCurrentCell;

    int currentParticle = gpu_cellVector[gpu_cellStartIndex[currentCell] + currentParticleIndex];
    int neighborParticle = gpu_cellVector[gpu_cellStartIndex[neighborCell] + neighborParticleIndex];

    if(currentParticle < neighborParticle && gpu_particleMol[currentParticle] != gpu_particleMol[neighborParticle]) {
      // Check if they are within rcut
      distSq = 0;
      double dx = gpu_x[currentParticle] - gpu_x[neighborParticle];
      double dy = gpu_y[currentParticle] - gpu_y[neighborParticle];
      double dz = gpu_z[currentParticle] - gpu_z[neighborParticle];

      dx = min(fabs(dx), xAxes - fabs(dx));
      dy = min(fabs(dy), yAxes - fabs(dy));
      dz = min(fabs(dz), zAxes - fabs(dz));

      distSq = dx * dx + dy * dy + dz * dz;

      if((cutoff * cutoff) > distSq) {
        myCounter++;
      }
    }
  }

  int cacheIndex = threadIdx.x;
  cache[cacheIndex] = myCounter;
  __syncthreads();
  int i = blockDim.x/2;
  while(i != 0){
    if(cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex+i];
    __syncthreads();
    i /= 2;
  }
  if (cacheIndex == 0)
    numIntersPerCell[blockIdx.x] = cache[0];
}

__global__ void BoxInterGPU(int *gpu_cellStartIndex,
                            int *gpu_cellVector,
                            int *gpu_neighborList,
                            int numberOfCells,
                            int cellVectorCount,
                            double *gpu_x,
                            double *gpu_y,
                            double *gpu_z,
                            double xAxes,
                            double yAxes,
                            double zAxes,
                            bool electrostatic,
                            double *gpu_particleCharge,
                            int *gpu_particleKind,
                            int *gpu_particleMol,
                            double *gpu_REn,
                            double *gpu_LJEn,
                            double *gpu_sigmaSq,
                            double *gpu_epsilon_Cn,
                            double *gpu_n,
                            int *gpu_VDW_Kind,
                            int *gpu_isMartini,
                            int *gpu_count,
                            double *gpu_rCut,
                            double *gpu_rCutCoulomb,
                            double *gpu_rCutLow,
                            double *gpu_rOn,
                            double *gpu_alpha,
                            int *gpu_ewald,
                            double *gpu_diElectric_1,
                            int *gpu_nonOrth,
                            double *gpu_cell_x,
                            double *gpu_cell_y,
                            double *gpu_cell_z,
                            double *gpu_Invcell_x,
                            double *gpu_Invcell_y,
                            double *gpu_Invcell_z,
                            bool sc_coul,
                            double sc_sigma_6,
                            double sc_alpha,
                            uint sc_power,
                            double *gpu_rMin,
                            double *gpu_rMaxSq,
                            double *gpu_expConst,
                            int *gpu_molIndex,
                            int *gpu_kindIndex,
                            double *gpu_lambdaVDW,
                            double *gpu_lambdaCoulomb,
                            bool *gpu_isFraction,
                            int box)
{
  double distSq;
  double qi_qj_fact;
  double qqFact = 167000.0;
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  gpu_REn[threadID] = 0.0;
  gpu_LJEn[threadID] = 0.0;
  double lambdaVDW = 0.0, lambdaCoulomb = 0.0;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);

  int currentCell = blockIdx.x / 27;
  int nCellIndex = blockIdx.x;
  int neighborCell = gpu_neighborList[nCellIndex];

  // calculate number of particles inside neighbor Cell
  int particlesInsideCurrentCell, particlesInsideNeighboringCells;
  int endIndex = neighborCell != numberOfCells - 1 ?
                 gpu_cellStartIndex[neighborCell + 1] : cellVectorCount;
  particlesInsideNeighboringCells = endIndex - gpu_cellStartIndex[neighborCell];

  // Calculate number of particles inside current Cell
  endIndex = currentCell != numberOfCells - 1 ?
             gpu_cellStartIndex[currentCell + 1] : cellVectorCount;
  particlesInsideCurrentCell = endIndex - gpu_cellStartIndex[currentCell];

  // total number of pairs
  int numberOfPairs = particlesInsideCurrentCell * particlesInsideNeighboringCells;

  for(int pairIndex = threadIdx.x; pairIndex < numberOfPairs; pairIndex += blockDim.x) {
    int neighborParticleIndex = pairIndex / particlesInsideCurrentCell;
    int currentParticleIndex = pairIndex % particlesInsideCurrentCell;

    int currentParticle = gpu_cellVector[gpu_cellStartIndex[currentCell] + currentParticleIndex];
    int neighborParticle = gpu_cellVector[gpu_cellStartIndex[neighborCell] + neighborParticleIndex];

    if(currentParticle < neighborParticle && gpu_particleMol[currentParticle] != gpu_particleMol[neighborParticle]) {
      // Check if they are within rcut
      distSq = 0;
      double dx = gpu_x[currentParticle] - gpu_x[neighborParticle];
      double dy = gpu_y[currentParticle] - gpu_y[neighborParticle];
      double dz = gpu_z[currentParticle] - gpu_z[neighborParticle];

      dx = min(fabs(dx), xAxes - fabs(dx));
      dy = min(fabs(dy), yAxes - fabs(dy));
      dz = min(fabs(dz), zAxes - fabs(dz));

      distSq = dx * dx + dy * dy + dz * dz;

      if((cutoff * cutoff) > distSq) {
        double cA = gpu_particleCharge[currentParticle];
        double cB = gpu_particleCharge[neighborParticle];
        int kA = gpu_particleKind[currentParticle];
        int kB = gpu_particleKind[neighborParticle];
        int mA = gpu_particleMol[currentParticle];
        int mB = gpu_particleMol[neighborParticle];

        lambdaVDW = DeviceGetLambdaVDW(mA, kA, mB, kB, box, gpu_isFraction,
                                       gpu_molIndex, gpu_kindIndex, gpu_lambdaVDW);

        if(electrostatic) {
          qi_qj_fact = cA * cB * qqFact;
          lambdaCoulomb = DeviceGetLambdaCoulomb(mA, kA, mB, kB, box,
                                                 gpu_isFraction, gpu_molIndex,
                                                 gpu_kindIndex, gpu_lambdaCoulomb);
          gpu_REn[threadID] += CalcCoulombGPU(distSq, kA, kB,
                                              qi_qj_fact, gpu_rCutLow[0],
                                              gpu_ewald[0], gpu_VDW_Kind[0],
                                              gpu_alpha[box],
                                              gpu_rCutCoulomb[box],
                                              gpu_isMartini[0],
                                              gpu_diElectric_1[0],
                                              lambdaCoulomb,
                                              sc_coul,
                                              sc_sigma_6,
                                              sc_alpha,
                                              sc_power,
                                              gpu_sigmaSq[box],
                                              gpu_count[0]);
        }
        gpu_LJEn[threadID] += CalcEnGPU(distSq, kA, kB,
                                        gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
                                        gpu_VDW_Kind[0], gpu_isMartini[0],
                                        gpu_rCut[0], gpu_rOn[0], gpu_count[0], lambdaVDW,
                                        sc_sigma_6, sc_alpha, sc_power, gpu_rMin,
                                        gpu_rMaxSq, gpu_expConst);
      }
    }
  }
}

__global__ void BoxInterGPUFlattened(int *gpu_cellStartIndex,
                            int *gpu_cellVector,
                            int *gpu_neighborList,
                            int numberOfCells,
                            int cellVectorCount,
                            double *gpu_x,
                            double *gpu_y,
                            double *gpu_z,
                            double xAxes,
                            double yAxes,
                            double zAxes,
                            bool electrostatic,
                            double *gpu_particleCharge,
                            int *gpu_particleKind,
                            int *gpu_particleMol,
                            double *gpu_REn,
                            double *gpu_LJEn,
                            double *gpu_sigmaSq,
                            double *gpu_epsilon_Cn,
                            double *gpu_n,
                            int *gpu_VDW_Kind,
                            int *gpu_isMartini,
                            int *gpu_count,
                            double *gpu_rCut,
                            double *gpu_rCutCoulomb,
                            double *gpu_rCutLow,
                            double *gpu_rOn,
                            double *gpu_alpha,
                            int *gpu_ewald,
                            double *gpu_diElectric_1,
                            int *gpu_nonOrth,
                            double *gpu_cell_x,
                            double *gpu_cell_y,
                            double *gpu_cell_z,
                            double *gpu_Invcell_x,
                            double *gpu_Invcell_y,
                            double *gpu_Invcell_z,
                            bool sc_coul,
                            double sc_sigma_6,
                            double sc_alpha,
                            uint sc_power,
                            double *gpu_rMin,
                            double *gpu_rMaxSq,
                            double *gpu_expConst,
                            int *gpu_molIndex,
                            int *gpu_kindIndex,
                            double *gpu_lambdaVDW,
                            double *gpu_lambdaCoulomb,
                            bool *gpu_isFraction,
                            int box,
                            uint * numIntersPerCell, 
                            double *flatgpu_REn,
                            double *flatgpu_LJEn,
                            int * flatIndexREn,
                            int * flatIndexLJEn
                          )
{
  double distSq;
  double qi_qj_fact;
  double qqFact = 167000.0;
  int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  gpu_REn[threadID] = 0.0;
  gpu_LJEn[threadID] = 0.0;
  double lambdaVDW = 0.0, lambdaCoulomb = 0.0;
  double cutoff = fmax(gpu_rCut[0], gpu_rCutCoulomb[box]);

  int currentCell = blockIdx.x / 27;
  int nCellIndex = blockIdx.x;
  int neighborCell = gpu_neighborList[nCellIndex];

  // calculate number of particles inside neighbor Cell
  int particlesInsideCurrentCell, particlesInsideNeighboringCells;
  int endIndex = neighborCell != numberOfCells - 1 ?
  gpu_cellStartIndex[neighborCell + 1] : cellVectorCount;
  particlesInsideNeighboringCells = endIndex - gpu_cellStartIndex[neighborCell];

  // Calculate number of particles inside current Cell
  endIndex = currentCell != numberOfCells - 1 ?
  gpu_cellStartIndex[currentCell + 1] : cellVectorCount;
  particlesInsideCurrentCell = endIndex - gpu_cellStartIndex[currentCell];

  // total number of pairs aka entries in our NxM matrix or r_ij
  int numberOfPairs = particlesInsideCurrentCell * particlesInsideNeighboringCells;

  for(int pairIndex = threadIdx.x; pairIndex < numberOfPairs; pairIndex += blockDim.x) {
  int neighborParticleIndex = pairIndex / particlesInsideCurrentCell;
  int currentParticleIndex = pairIndex % particlesInsideCurrentCell;

  int currentParticle = gpu_cellVector[gpu_cellStartIndex[currentCell] + currentParticleIndex];
  int neighborParticle = gpu_cellVector[gpu_cellStartIndex[neighborCell] + neighborParticleIndex];

    if(currentParticle < neighborParticle && gpu_particleMol[currentParticle] != gpu_particleMol[neighborParticle]) {
      // Check if they are within rcut
      distSq = 0;
      double dx = gpu_x[currentParticle] - gpu_x[neighborParticle];
      double dy = gpu_y[currentParticle] - gpu_y[neighborParticle];
      double dz = gpu_z[currentParticle] - gpu_z[neighborParticle];

      dx = min(fabs(dx), xAxes - fabs(dx));
      dy = min(fabs(dy), yAxes - fabs(dy));
      dz = min(fabs(dz), zAxes - fabs(dz));

      distSq = dx * dx + dy * dy + dz * dz;

      if((cutoff * cutoff) > distSq) {
        double cA = gpu_particleCharge[currentParticle];
        double cB = gpu_particleCharge[neighborParticle];
        int kA = gpu_particleKind[currentParticle];
        int kB = gpu_particleKind[neighborParticle];
        int mA = gpu_particleMol[currentParticle];
        int mB = gpu_particleMol[neighborParticle];

        lambdaVDW = DeviceGetLambdaVDW(mA, kA, mB, kB, box, gpu_isFraction,
                    gpu_molIndex, gpu_kindIndex, gpu_lambdaVDW);

        if(electrostatic) {
          qi_qj_fact = cA * cB * qqFact;
          lambdaCoulomb = DeviceGetLambdaCoulomb(mA, kA, mB, kB, box,
                                gpu_isFraction, gpu_molIndex,
                                gpu_kindIndex, gpu_lambdaCoulomb);
          gpu_REn[threadID] += CalcCoulombGPU(distSq, kA, kB,
                              qi_qj_fact, gpu_rCutLow[0],
                              gpu_ewald[0], gpu_VDW_Kind[0],
                              gpu_alpha[box],
                              gpu_rCutCoulomb[box],
                              gpu_isMartini[0],
                              gpu_diElectric_1[0],
                              lambdaCoulomb,
                              sc_coul,
                              sc_sigma_6,
                              sc_alpha,
                              sc_power,
                              gpu_sigmaSq[box],
                              gpu_count[0]);
              flatgpu_REn[atomicAdd(flatIndexREn, 1)] = CalcCoulombGPU( distSq, kA, kB,
                                                        qi_qj_fact, gpu_rCutLow[0],
                                                        gpu_ewald[0], gpu_VDW_Kind[0],
                                                        gpu_alpha[box],
                                                        gpu_rCutCoulomb[box],
                                                        gpu_isMartini[0],
                                                        gpu_diElectric_1[0],
                                                        lambdaCoulomb,
                                                        sc_coul,
                                                        sc_sigma_6,
                                                        sc_alpha,
                                                        sc_power,
                                                        gpu_sigmaSq[box],
                                                        gpu_count[0]);
                                                       
            
        }
        gpu_LJEn[threadID] += CalcEnGPU(distSq, kA, kB,
                      gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
                      gpu_VDW_Kind[0], gpu_isMartini[0],
                      gpu_rCut[0], gpu_rOn[0], gpu_count[0], lambdaVDW,
                      sc_sigma_6, sc_alpha, sc_power, gpu_rMin,
                      gpu_rMaxSq, gpu_expConst);
          flatgpu_LJEn[atomicAdd(flatIndexLJEn, 1)] = CalcEnGPU(distSq, kA, kB,
                      gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
                      gpu_VDW_Kind[0], gpu_isMartini[0],
                      gpu_rCut[0], gpu_rOn[0], gpu_count[0], lambdaVDW,
                      sc_sigma_6, sc_alpha, sc_power, gpu_rMin,
                      gpu_rMaxSq, gpu_expConst);
      }
    }
    //flatThreadID += blockDim.x;
  }
}

__device__ double CalcCoulombGPU(double distSq,
                                 int kind1,
                                 int kind2,
                                 double qi_qj_fact,
                                 double gpu_rCutLow,
                                 int gpu_ewald,
                                 int gpu_VDW_Kind,
                                 double gpu_alpha,
                                 double gpu_rCutCoulomb,
                                 int gpu_isMartini,
                                 double gpu_diElectric_1,
                                 double gpu_lambdaCoulomb,
                                 bool sc_coul,
                                 double sc_sigma_6,
                                 double sc_alpha,
                                 uint sc_power,
                                 double gpu_sigmaSq,
                                 int gpu_count)
{
  if((gpu_rCutCoulomb * gpu_rCutCoulomb) < distSq) {
    return 0.0;
  }

  int index = FlatIndexGPU(kind1, kind2, gpu_count);
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND) {
    return CalcCoulombParticleGPU(distSq, qi_qj_fact, gpu_ewald, gpu_alpha,
                                  gpu_lambdaCoulomb, sc_coul, sc_sigma_6,
                                  sc_alpha, sc_power, gpu_sigmaSq);
  } else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND) {
    return CalcCoulombShiftGPU(distSq, qi_qj_fact, gpu_ewald, gpu_alpha,
                               gpu_rCutCoulomb, gpu_lambdaCoulomb, sc_coul,
                               sc_sigma_6, sc_alpha, sc_power, gpu_sigmaSq);
  } else if(gpu_VDW_Kind == GPU_VDW_EXP6_KIND) {
    return CalcCoulombExp6GPU(distSq, qi_qj_fact, gpu_ewald, gpu_alpha,
                              gpu_lambdaCoulomb, sc_coul, sc_sigma_6, sc_alpha,
                              sc_power, gpu_sigmaSq);
  } else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini) {
    return CalcCoulombSwitchMartiniGPU(distSq, qi_qj_fact, gpu_ewald, gpu_alpha,
                                       gpu_rCutCoulomb, gpu_diElectric_1,
                                       gpu_lambdaCoulomb, sc_coul, sc_sigma_6,
                                       sc_alpha, sc_power, gpu_sigmaSq);
  } else
    return CalcCoulombSwitchGPU(distSq, qi_qj_fact, gpu_alpha, gpu_ewald,
                                gpu_rCutCoulomb, gpu_lambdaCoulomb,
                                sc_coul, sc_sigma_6, sc_alpha, sc_power,
                                gpu_sigmaSq);
}

__device__ double CalcEnGPU(double distSq, int kind1, int kind2,
                            double *gpu_sigmaSq, double *gpu_n,
                            double *gpu_epsilon_Cn, int gpu_VDW_Kind,
                            int gpu_isMartini, double gpu_rCut, double gpu_rOn,
                            int gpu_count, double gpu_lambdaVDW,
                            double sc_sigma_6, double sc_alpha, uint sc_power,
                            double *gpu_rMin, double *gpu_rMaxSq,
                            double *gpu_expConst)
{
  if((gpu_rCut * gpu_rCut) < distSq) {
    return 0.0;
  }

  int index = FlatIndexGPU(kind1, kind2, gpu_count);
  if(gpu_VDW_Kind == GPU_VDW_STD_KIND) {
    return CalcEnParticleGPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
                             gpu_lambdaVDW, sc_sigma_6, sc_alpha, sc_power);
  } else if(gpu_VDW_Kind == GPU_VDW_SHIFT_KIND) {
    return CalcEnShiftGPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
                          gpu_rCut, gpu_lambdaVDW, sc_sigma_6, sc_alpha,
                          sc_power);
  } else if(gpu_VDW_Kind == GPU_VDW_EXP6_KIND) {
    return CalcEnExp6GPU(distSq, index, gpu_sigmaSq[index], gpu_n[index],
                         gpu_lambdaVDW, sc_sigma_6,
                         sc_alpha, sc_power, gpu_rMin[index],
                         gpu_rMaxSq[index], gpu_expConst[index]);
  } else if(gpu_VDW_Kind == GPU_VDW_SWITCH_KIND && gpu_isMartini) {
    return CalcEnSwitchMartiniGPU(distSq, index, gpu_sigmaSq, gpu_n,
                                  gpu_epsilon_Cn, gpu_rCut, gpu_rOn,
                                  gpu_lambdaVDW, sc_sigma_6, sc_alpha,
                                  sc_power);
  } else
    return CalcEnSwitchGPU(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn,
                           gpu_rCut, gpu_rOn, gpu_lambdaVDW, sc_sigma_6,
                           sc_alpha, sc_power);
}

//ElectroStatic Calculation
//**************************************************************//
__device__ double CalcCoulombParticleGPU(double distSq, double qi_qj_fact,
    double gpu_ewald, double gpu_alpha,
    double gpu_lambdaCoulomb, bool sc_coul,
    double sc_sigma_6, double sc_alpha,
    uint sc_power, double gpu_sigmaSq)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombParticleGPUNoLambda(distSq, qi_qj_fact, gpu_ewald, gpu_alpha);
  }
  if(sc_coul) {
    double sigma6 = gpu_sigmaSq * gpu_sigmaSq * gpu_sigmaSq;
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 * dist6;
    double softRsq = pow(softDist6, 1.0 / 3.0);
    return gpu_lambdaCoulomb * CalcCoulombParticleGPUNoLambda(softRsq, qi_qj_fact, gpu_ewald, gpu_alpha);
  } else {
    return gpu_lambdaCoulomb * CalcCoulombParticleGPUNoLambda(distSq, qi_qj_fact, gpu_ewald, gpu_alpha);
  }
}

__device__ double CalcCoulombParticleGPUNoLambda(double distSq,
    double qi_qj_fact,
    double gpu_ewald,
    double gpu_alpha)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double value = gpu_alpha * dist;
    return qi_qj_fact * erfc(value) / dist;
  } else {
    double dist = sqrt(distSq);
    return qi_qj_fact / dist;
  }
}

__device__ double CalcCoulombShiftGPU(double distSq, double qi_qj_fact,
                                      int gpu_ewald, double gpu_alpha,
                                      double gpu_rCut, double gpu_lambdaCoulomb,
                                      bool sc_coul, double sc_sigma_6,
                                      double sc_alpha, uint sc_power,
                                      double gpu_sigmaSq)
{

  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombShiftGPUNoLambda(distSq, qi_qj_fact, gpu_ewald, gpu_alpha,
                                       gpu_rCut);
  }

  if(sc_coul) {
    double sigma6 = gpu_sigmaSq * gpu_sigmaSq * gpu_sigmaSq;
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 * dist6;
    double softRsq = pow(softDist6, (double)1.0 / 3.0);
    return gpu_lambdaCoulomb * CalcCoulombShiftGPUNoLambda(softRsq, qi_qj_fact,
           gpu_ewald, gpu_alpha,
           gpu_rCut);
  } else {
    return gpu_lambdaCoulomb * CalcCoulombShiftGPUNoLambda(distSq, qi_qj_fact,
           gpu_ewald, gpu_alpha,
           gpu_rCut);
  }
}

__device__ double CalcCoulombShiftGPUNoLambda(double distSq, double qi_qj_fact,
    int gpu_ewald, double gpu_alpha,
    double gpu_rCut)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double value = gpu_alpha * dist;
    return qi_qj_fact * (1 - erf(value)) / dist;
  } else {
    double dist = sqrt(distSq);
    return qi_qj_fact * (1.0 / dist - 1.0 / gpu_rCut);
  }
}

__device__ double CalcCoulombExp6GPU(double distSq, double qi_qj_fact,
                                     int gpu_ewald, double gpu_alpha,
                                     double gpu_lambdaCoulomb, bool sc_coul,
                                     double sc_sigma_6, double sc_alpha,
                                     uint sc_power, double gpu_sigmaSq)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombExp6GPUNoLambda(distSq, qi_qj_fact, gpu_ewald, gpu_alpha);
  }

  if(sc_coul) {
    double sigma6 = gpu_sigmaSq * gpu_sigmaSq * gpu_sigmaSq;
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb),
                                       (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 * dist6;
    double softRsq = pow(softDist6, (double)1.0 / 3.0);
    return gpu_lambdaCoulomb * CalcCoulombExp6GPUNoLambda(softRsq, qi_qj_fact,
           gpu_ewald, gpu_alpha);
  } else {
    return gpu_lambdaCoulomb * CalcCoulombExp6GPUNoLambda(distSq, qi_qj_fact,
           gpu_ewald, gpu_alpha);
  }
}

__device__ double CalcCoulombExp6GPUNoLambda(double distSq, double qi_qj_fact,
    int gpu_ewald, double gpu_alpha)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double val = gpu_alpha * dist;
    return qi_qj_fact * erfc(val) / dist;
  } else {
    double dist = sqrt(distSq);
    return qi_qj_fact / dist;
  }
}

__device__ double CalcCoulombSwitchMartiniGPU(double distSq, double qi_qj_fact,
    int gpu_ewald, double gpu_alpha,
    double gpu_rCut,
    double gpu_diElectric_1,
    double gpu_lambdaCoulomb,
    bool sc_coul, double sc_sigma_6,
    double sc_alpha, uint sc_power,
    double gpu_sigmaSq)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombSwitchMartiniGPUNoLambda(distSq, qi_qj_fact, gpu_ewald, gpu_alpha, gpu_rCut, gpu_diElectric_1);
  }

  if(sc_coul) {
    double sigma6 = gpu_sigmaSq * gpu_sigmaSq * gpu_sigmaSq;
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 * dist6;
    double softRsq = pow(softDist6, 1.0 / 3.0);
    return gpu_lambdaCoulomb * CalcCoulombSwitchMartiniGPUNoLambda(softRsq, qi_qj_fact, gpu_ewald, gpu_alpha, gpu_rCut, gpu_diElectric_1);
  } else {
    return gpu_lambdaCoulomb * CalcCoulombSwitchMartiniGPUNoLambda(distSq, qi_qj_fact, gpu_ewald, gpu_alpha, gpu_rCut, gpu_diElectric_1);
  }
}

__device__ double CalcCoulombSwitchMartiniGPUNoLambda(double distSq,
    double qi_qj_fact,
    int gpu_ewald,
    double gpu_alpha,
    double gpu_rCut,
    double gpu_diElectric_1)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double value = gpu_alpha * dist;
    return qi_qj_fact * (1 - erf(value)) / dist;
  } else {
    // in Martini, the Coulomb switching distance is zero, so we will have
    // sqrt(distSq) - rOnCoul =  sqrt(distSq)
    double dist = sqrt(distSq);
    double rij_ronCoul_3 = dist * distSq;
    double rij_ronCoul_4 = distSq * distSq;

    double A1 = 1.0 * (-(1.0 + 4) * gpu_rCut) / (pow(gpu_rCut, 1.0 + 2) *
                pow(gpu_rCut, 2));
    double B1 = -1.0 * (-(1.0 + 3) * gpu_rCut) / (pow(gpu_rCut, 1.0 + 2) *
                pow(gpu_rCut, 3));
    double C1 = 1.0 / pow(gpu_rCut, 1.0) - A1 / 3.0 * pow(gpu_rCut, 3) -
                B1 / 4.0 * pow(gpu_rCut, 4);

    double coul = -(A1 / 3.0) * rij_ronCoul_3 - (B1 / 4.0) * rij_ronCoul_4 - C1;
    return qi_qj_fact * gpu_diElectric_1 * (1.0 / dist + coul);
  }
}

__device__ double CalcCoulombSwitchGPU(double distSq, double qi_qj_fact,
                                       double gpu_alpha, int gpu_ewald,
                                       double gpu_rCut,
                                       double gpu_lambdaCoulomb, bool sc_coul,
                                       double sc_sigma_6, double sc_alpha,
                                       uint sc_power, double gpu_sigmaSq)
{
  if(gpu_lambdaCoulomb >= 0.999999) {
    return CalcCoulombSwitchGPUNoLambda(distSq, qi_qj_fact, gpu_ewald, gpu_alpha, gpu_rCut);
  }

  if(sc_coul) {
    double sigma6 = gpu_sigmaSq * gpu_sigmaSq * gpu_sigmaSq;
    sigma6 = max(sigma6, sc_sigma_6);
    double dist6 = distSq * distSq * distSq;
    double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaCoulomb), (double)sc_power);
    double softDist6 = lambdaCoef * sigma6 * dist6;
    double softRsq = pow(softDist6, 1.0 / 3.0);
    return gpu_lambdaCoulomb * CalcCoulombSwitchGPUNoLambda(softRsq, qi_qj_fact, gpu_ewald, gpu_alpha, gpu_rCut);
  } else {
    return gpu_lambdaCoulomb * CalcCoulombSwitchGPUNoLambda(distSq, qi_qj_fact, gpu_ewald, gpu_alpha, gpu_rCut);
  }
}

__device__ double CalcCoulombSwitchGPUNoLambda(double distSq, double qi_qj_fact,
    double gpu_alpha, int gpu_ewald,
    double gpu_rCut)
{
  if(gpu_ewald) {
    double dist = sqrt(distSq);
    double value = gpu_alpha * dist;
    return qi_qj_fact * (1 - erf(value)) / dist;
  } else {
    double rCutSq = gpu_rCut * gpu_rCut;
    double dist = sqrt(distSq);
    double switchVal = distSq / rCutSq - 1.0;
    switchVal *= switchVal;
    return qi_qj_fact * switchVal / dist;
  }
}

//VDW Calculation
//**************************************************************//
__device__ double CalcEnParticleGPU(double distSq, int index,
                                    double *gpu_sigmaSq, double *gpu_n,
                                    double *gpu_epsilon_Cn,
                                    double gpu_lambdaVDW,
                                    double sc_sigma_6,
                                    double sc_alpha,
                                    uint sc_power)
{
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcEnParticleGPUNoLambda(distSq, index, gpu_sigmaSq, gpu_n, gpu_epsilon_Cn);
  }

  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, 1.0 / 3.0);

  return gpu_lambdaVDW * CalcEnParticleGPUNoLambda(softRsq, index, gpu_sigmaSq,
         gpu_n, gpu_epsilon_Cn);
}

__device__ double CalcEnParticleGPUNoLambda(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn)
{
  double rRat2 = gpu_sigmaSq[index] / distSq;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);
  return gpu_epsilon_Cn[index] * (repulse - attract);
}

__device__ double CalcEnShiftGPU(double distSq, int index, double *gpu_sigmaSq,
                                 double *gpu_n, double *gpu_epsilon_Cn,
                                 double gpu_rCut,
                                 double gpu_lambdaVDW,
                                 double sc_sigma_6,
                                 double sc_alpha,
                                 uint sc_power)
{
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcEnShiftGPUNoLambda(distSq, index, gpu_sigmaSq, gpu_n,
                                  gpu_epsilon_Cn, gpu_rCut);
  }

  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);

  return gpu_lambdaVDW * CalcEnShiftGPUNoLambda(softRsq, index, gpu_sigmaSq,
         gpu_n, gpu_epsilon_Cn,
         gpu_rCut);
}

__device__ double CalcEnShiftGPUNoLambda(double distSq, int index,
    double *gpu_sigmaSq,
    double *gpu_n, double *gpu_epsilon_Cn,
    double gpu_rCut)
{
  double rRat2 = gpu_sigmaSq[index] / distSq;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;
  double repulse = pow(rRat2, gpu_n[index] / 2.0);

  double shiftRRat2 = gpu_sigmaSq[index] / (gpu_rCut * gpu_rCut);
  double shiftRRat4 = shiftRRat2 * shiftRRat2;
  double shiftAttract = shiftRRat4 * shiftRRat2;
  double shiftRepulse = pow(shiftRRat2, gpu_n[index] / 2.0);
  double shiftConst = gpu_epsilon_Cn[index] * (shiftRepulse - shiftAttract);

  return (gpu_epsilon_Cn[index] * (repulse - attract) - shiftConst);
}

__device__ double CalcEnExp6GPU(double distSq, int index, double gpu_sigmaSq,
                                double gpu_n, double gpu_lambdaVDW,
                                double sc_sigma_6, double sc_alpha,
                                uint sc_power, double gpu_rMin,
                                double gpu_rMaxSq, double gpu_expConst)
{
  if(distSq < gpu_rMaxSq) {
    return num::BIGNUM;
  }
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcEnExp6GPUNoLambda(distSq, gpu_n, gpu_rMin, gpu_expConst);
  }
  double sigma6 = gpu_sigmaSq * gpu_sigmaSq * gpu_sigmaSq;
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);

  return gpu_lambdaVDW * CalcEnExp6GPUNoLambda(softRsq,  gpu_n, gpu_rMin,
         gpu_expConst);
}

__device__ double CalcEnExp6GPUNoLambda(double distSq, double gpu_n,
                                        double gpu_rMin, double gpu_expConst)
{
  double dist = sqrt(distSq);
  double rRat = gpu_rMin / dist;
  double rRat2 = rRat * rRat;
  double attract = rRat2 * rRat2 * rRat2;

  uint alph_ij = gpu_n;
  double repulse = (6.0 / alph_ij) * exp(alph_ij * (1.0 - dist / gpu_rMin));
  return gpu_expConst * (repulse - attract);
}

__device__ double CalcEnSwitchMartiniGPU(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn,
    double gpu_rCut, double gpu_rOn,
    double gpu_lambdaVDW,
    double sc_sigma_6,
    double sc_alpha,
    uint sc_power)
{
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcEnSwitchMartiniGPUNoLambda(distSq, index, gpu_sigmaSq, gpu_n,
                                          gpu_epsilon_Cn, gpu_rCut, gpu_rOn);
  }

  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);

  return gpu_lambdaVDW * CalcEnSwitchMartiniGPUNoLambda(softRsq, index,
         gpu_sigmaSq, gpu_n,
         gpu_epsilon_Cn,
         gpu_rCut, gpu_rOn);
}

__device__ double CalcEnSwitchMartiniGPUNoLambda(double distSq, int index,
    double *gpu_sigmaSq,
    double *gpu_n,
    double *gpu_epsilon_Cn,
    double gpu_rCut,
    double gpu_rOn)
{
  double r_2 = 1.0 / distSq;
  double r_4 = r_2 * r_2;
  double r_6 = r_4 * r_2;
  double r_n = pow(r_2, gpu_n[index] / 2.0);

  double rij_ron = sqrt(distSq) - gpu_rOn;
  double rij_ron_2 = rij_ron * rij_ron;
  double rij_ron_3 = rij_ron_2 * rij_ron;
  double rij_ron_4 = rij_ron_2 * rij_ron_2;

  double pn = gpu_n[index];
  double An = pn * ((pn + 1) * gpu_rOn - (pn + 4) * gpu_rCut) /
              (pow(gpu_rCut, pn + 2) * pow(gpu_rCut - gpu_rOn, 2));
  double Bn = -pn * ((pn + 1) * gpu_rOn - (pn + 3) * gpu_rCut) /
              (pow(gpu_rCut, pn + 2) * pow(gpu_rCut - gpu_rOn, 3));
  double Cn = 1.0 / pow(gpu_rCut, pn) - An / 3.0 * pow(gpu_rCut - gpu_rOn, 3) -
              Bn / 4.0 * pow(gpu_rCut - gpu_rOn, 4);

  double A6 = 6.0 * ((6.0 + 1) * gpu_rOn - (6.0 + 4) * gpu_rCut) /
              (pow(gpu_rCut, 6.0 + 2) * pow(gpu_rCut - gpu_rOn, 2));
  double B6 = -6.0 * ((6.0 + 1) * gpu_rOn - (6.0 + 3) * gpu_rCut) /
              (pow(gpu_rCut, 6.0 + 2) * pow(gpu_rCut - gpu_rOn, 3));
  double C6 = 1.0 / pow(gpu_rCut, 6.0) - A6 / 3.0 * pow(gpu_rCut - gpu_rOn, 3) -
              B6 / 4.0 * pow(gpu_rCut - gpu_rOn, 4);

  double shifttempRep = -(An / 3.0) * rij_ron_3 -
                        (Bn / 4.0) * rij_ron_4 - Cn;
  double shifttempAtt = -(A6 / 3.0) * rij_ron_3 - (B6 / 4.0) * rij_ron_4 - C6;

  const double shiftRep = ( distSq > gpu_rOn * gpu_rOn ? shifttempRep : -Cn);
  const double shiftAtt = ( distSq > gpu_rOn * gpu_rOn ? shifttempAtt : -C6);

  double sig6 = pow(gpu_sigmaSq[index], 3);
  double sign = pow(gpu_sigmaSq[index], pn / 2);
  double Eij = gpu_epsilon_Cn[index] * (sign * (r_n + shiftRep) -
                                        sig6 * (r_6 + shiftAtt));
  return Eij;
}

__device__ double CalcEnSwitchGPU(double distSq, int index, double *gpu_sigmaSq,
                                  double *gpu_n, double *gpu_epsilon_Cn,
                                  double gpu_rCut, double gpu_rOn,
                                  double gpu_lambdaVDW, double sc_sigma_6,
                                  double sc_alpha, uint sc_power)
{
  if(gpu_lambdaVDW >= 0.999999) {
    return CalcEnSwitchGPUNoLambda(distSq, index, gpu_sigmaSq, gpu_n,
                                   gpu_epsilon_Cn, gpu_rCut, gpu_rOn);
  }
  double sigma6 = gpu_sigmaSq[index] * gpu_sigmaSq[index] * gpu_sigmaSq[index];
  sigma6 = max(sigma6, sc_sigma_6);
  double dist6 = distSq * distSq * distSq;
  double lambdaCoef = sc_alpha * pow((1.0 - gpu_lambdaVDW), (double)sc_power);
  double softDist6 = lambdaCoef * sigma6 + dist6;
  double softRsq = pow(softDist6, (double)1.0 / 3.0);

  return gpu_lambdaVDW * CalcEnSwitchGPUNoLambda(softRsq, index, gpu_sigmaSq,
         gpu_n, gpu_epsilon_Cn,
         gpu_rCut, gpu_rOn);
}

__device__ double CalcEnSwitchGPUNoLambda(double distSq, int index,
    double *gpu_sigmaSq, double *gpu_n,
    double *gpu_epsilon_Cn,
    double gpu_rCut, double gpu_rOn)
{
  double rCutSq = gpu_rCut * gpu_rCut;
  double rOnSq = gpu_rOn * gpu_rOn;

  double rCutSq_rijSq = rCutSq  - distSq;
  double rCutSq_rijSq_Sq = rCutSq_rijSq * rCutSq_rijSq;

  double rRat2 = gpu_sigmaSq[index] / distSq;
  double rRat4 = rRat2 * rRat2;
  double attract = rRat4 * rRat2;

  double repulse = pow(rRat2, gpu_n[index] / 2.0);

  double factor1 = rCutSq - 3 * rOnSq;
  double factor2 = pow((rCutSq - rOnSq), -3);
  double fE = rCutSq_rijSq_Sq * factor2 * (factor1 + 2 * distSq);

  const double factE = ( distSq > rOnSq ? fE : 1.0);

  return (gpu_epsilon_Cn[index] * (repulse - attract)) * factE;
}

struct Lock {
  int * mutex;
  Lock (void) {
    int state = 0;
    CUMALLOC((void**)&mutex, sizeof(int));
    cudaMemcpy(mutex, &state, sizeof(int), cudaMemcpyHostToDevice);
  }

  ~Lock (void){
    cudaFree(mutex);
  }


__device__ void lock( void ) {
  while( atomicCAS( mutex, 0, 1 ) != 0 );
  __threadfence();}
  
  __device__ void unlock( void ) {
    __threadfence();
    atomicExch( mutex, 0 );
  }
};

#endif
